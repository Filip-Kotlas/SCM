#include "hip/hip_runtime.h"
#include "binaryIO.h"
//#include "crsmatrix.h"
#include "vdop.h"

#include <algorithm>
#include <cassert>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>
using namespace std;

// ####################################################################
#include "crsmatrix_gpu.h"
#include "vdop_gpu.h"
#include <hipsparse.h>

//----------------------------------------------------------------------

CRS_Matrix_GPU::CRS_Matrix_GPU()
{
    
}

CRS_Matrix_GPU::CRS_Matrix_GPU(std::vector<int>    const& rowOffset, 
                   std::vector<int>    const& colIndices, 
                   std::vector<double> const& nnzValues)
                   :_nrows(static_cast<int>(size(rowOffset) - 1)), _ncols(_nrows), _nnz(static_cast<int>(size(nnzValues))) 
{
	setupMemory_GPU(rowOffset, colIndices, nnzValues);
}                   
// TODO
//   - write a private function setupMemory_GPU(rowOffset,colIndices,nnzValues)
//     that does the GPU memory initalization as in the constructor below
//   - call this method from both constructors
//   - maybe a third constructor CRS_Matrix_GPU(CRS_Matrix const&) and
//       * get rid of the (unnecessary) inheritance of CRS_Matrix_GPU from CRS_Matrix
//       * some members/functins from CRS_Matrix have to be added in this case.
//       * the constructor below will call generate a temp. matrix 
//         on CPU via CRS_Matrix(file) in the function body

void CRS_Matrix_GPU::setupMemory_GPU(std::vector<int> const& rowOffset, std::vector<int> const& colIndices, 
                                     std::vector<double> const& nnzValues) 
{
    cout << "IN  :: CRS_Matrix_GPU::setupMemory_GPU" << endl;
// 	https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuSPARSE/cg
	CHECK_CUBLAS( hipblasCreate(&_cublasHandle) )
    CHECK_CUSPARSE( hipsparseCreate(&_cusparseHandle) )
	
	// row offset  --> unified memory
	size_t nBytes;
	nBytes = (Nrows()+1)*sizeof(*_d_rowOffsets);      // int*4
	CHECK_CUDA( hipMalloc(&_d_rowOffsets, nBytes) )
	//CHECK_CUDA( hipMallocManaged(&_d_rowOffsets, nBytes) )
        //   hipMemcpyAsync slows down my code (cg)
    CHECK_CUDA( hipMemcpy(_d_rowOffsets, rowOffset.data(), nBytes, hipMemcpyDefault) );	

	// column indices --> unified memory
	nBytes = Nnz()*sizeof(*_d_colIndices);            // int*4
    CHECK_CUDA( hipMalloc(&_d_colIndices, nBytes) )
	//CHECK_CUDA( hipMallocManaged(&_d_colIndices, nBytes) )
    CHECK_CUDA( hipMemcpy(_d_colIndices, colIndices.data(), nBytes, hipMemcpyDefault) );	
   
    // non-zero entries --> unified memory
    nBytes = Nnz()*sizeof(*_d_values);                // real*8
    CHECK_CUDA( hipMalloc(&_d_values, nBytes) )
	//CHECK_CUDA( hipMallocManaged(&_d_values, nBytes) )
    CHECK_CUDA( hipMemcpy(_d_values, nnzValues.data(), nBytes, hipMemcpyDefault) );

    // combine everthing to one cuSPARSE matrix (CSR)
    CHECK_CUSPARSE( 
     hipsparseCreateCsr(&_matA, Nrows(), Ncols(), Nnz(), 
                       _d_rowOffsets, _d_colIndices, _d_values,
                       ROWOFFTYPE, COLIDXTYPE, baseIdx, REALTYPE) )
    //------------------------------------------------------------------
    // allocate some buffer for cuSPARSE
       // aux. vectors device
    Vec d_X(Nrows());
    Vec d_B(Nrows());	
	cout << "MID :: CRS_Matrix_GPU::setupMemory_GPU" << endl;
    cout << "Nrows(): " << Nrows() << ", Ncols(): " << Ncols() << std::endl; 

    double alpha=1.0;
    double beta =0.0;
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                        _cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, _matA, d_X.sphandler(), &beta, d_B.sphandler(), REALTYPE,
                        ALGTYPE, &_bufferSizeMV) )
    CHECK_CUDA( hipMalloc(&_dbufferMV, _bufferSizeMV) )

    CHECK_CUSPARSE( hipsparseSpMV(
                        _cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, _matA, d_X.sphandler(), &beta, d_B.sphandler(), HIP_R_64F,
                        ALGTYPE, _dbufferMV) )  
    //------------------------------------------------------------------
    hipDeviceSynchronize();
    cout << "OUT :: CRS_Matrix_GPU::setupMemory_GPU" << endl; 
}

CRS_Matrix_GPU::CRS_Matrix_GPU(CRS_Matrix const& matrix)
:_nrows(matrix.Nrows()), _ncols(matrix.Ncols()), _nnz(matrix.Nnz())
{
    setupMemory_GPU( matrix.get_RowOffset(), matrix.get_ColumnIndices(), matrix.get_NnzValues());
}

CRS_Matrix_GPU::CRS_Matrix_GPU( BisectIntDirichlet const &matrix)
:_nrows(matrix.Nrows()), _ncols(matrix.Ncols())
{
    vector<int> rowOffset;
    vector<int> colIndices;
    vector<double> nnzValues;

    double EPS = 1e-8;
    int rowCount = 0;
    for( int i = 0; i < matrix.Nrows(); i++ )
    {
        for( int j = 0; j < matrix.Ncols(); j++ )
        {
            if( abs( matrix(i, j) ) > EPS )
            {
                nnzValues.emplace_back( matrix(i, j) );
                colIndices.emplace_back(j);
                rowCount++;
            }
        }
        rowOffset.emplace_back(rowCount);
    }
    _nnz = nnzValues.size();
    setupMemory_GPU( rowOffset, colIndices, nnzValues);
}

CRS_Matrix_GPU::CRS_Matrix_GPU(const std::string& file)
{
    CRS_Matrix temp(file);
    _nrows = temp.Nrows();
    _ncols = temp.Ncols();
    _nnz = temp.Nnz();
    setupMemory_GPU( temp.get_RowOffset(), temp.get_ColumnIndices(), temp.get_NnzValues() );
}

CRS_Matrix_GPU::~CRS_Matrix_GPU()
{
    hipblasDestroy(_cublasHandle);
    hipsparseDestroy(_cusparseHandle);
	hipsparseDestroySpMat(_matA);
// Free the Unified Memory
    hipFree(_dbufferMV);
    hipFree(_d_values); hipFree(_d_colIndices); hipFree(_d_rowOffsets);
}

void CRS_Matrix_GPU::Debug() const
{
//  ID points to first entry of row
//  no symmetry assumed
    cout << "\nMatrix  (" << _nrows << " x " << _ncols << "  with  nnz = " << Nnz() << ")\n";
    cout << _d_rowOffsets << "   " << _d_colIndices << "   " << _d_values << endl;

    for (int row = 0; row < _nrows; ++row)
    {
        cout << "Row " << row << " : ";
        int const id1 = _d_rowOffsets[row];
        int const id2 = _d_rowOffsets[row + 1];
        for (int j = id1; j < id2; ++j)
        {
            cout.setf(ios::right, ios::adjustfield);
            cout << "[" << setw(2) << _d_colIndices[j] << "]  " << setw(4) << _d_values[j] << "  ";
        }
        cout << endl;
    }
    return;
}


void CRS_Matrix_GPU::Mult(Vec &d_w, Vec const &d_u) const
{
	double const zero(0.0), one(1.0);
	CHECK_CUSPARSE( 
	hipsparseSpMV(_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &one, _matA, d_u.sphandler(), &zero, d_w.sphandler(),
                 REALTYPE, ALGTYPE, _dbufferMV) )	
}

void CRS_Matrix_GPU::MultT(Vec const &d_w, Vec &d_u) const
{
    double const zero(0.0), one(1.0);
	CHECK_CUSPARSE( 
	hipsparseSpMV(_cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                 &one, _matA, d_w.sphandler(), &zero, d_u.sphandler(),
                 REALTYPE, ALGTYPE, _dbufferMV) )
}

void CRS_Matrix_GPU::Defect(Vec &d_r, Vec const &d_f, Vec const &d_u) const
{
	//  f --> r
    d_r = d_f;
    // r = -K*u+r
 	double const minus_one(-1.0), one(1.0);  
 	//CHECK_CUSPARSE( hipsparseCreate(&_cusparseHandle) ) 
    CHECK_CUSPARSE( 
    hipsparseSpMV(_cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                 &minus_one, _matA, d_u.sphandler(), &one, d_r.sphandler(), 
                 REALTYPE, ALGTYPE, _dbufferMV) )   
}

void CRS_Matrix_GPU::GetDiag(Vec &d) const
{
    int blockSize = 256;
    int numBlocks = (Ncols() + blockSize - 1) / blockSize;
    ExtractDiagKernel<<<numBlocks, blockSize>>>( Ncols(), _d_rowOffsets, _d_colIndices, _d_values, d.data());
    hipDeviceSynchronize();
}

void CRS_Matrix_GPU::GetInvDiag(Vec &d) const
{
    int blockSize = 256;
    int numBlocks = (Ncols() + blockSize - 1) / blockSize;
    ExtractInverseDiagKernel<<<numBlocks, blockSize>>>( Ncols(), _d_rowOffsets, _d_colIndices, _d_values, d.data());
    hipDeviceSynchronize();
}

__global__ void ExtractDiagKernel(int n, const int* row_offsets, const int* col_indices, const double* values, double* diagonal) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n)
    {
        diagonal[row] = 0.0f; // Initialize diagonal element
        for(int i = row_offsets[row]; i < row_offsets[row + 1]; i++)
        {
            if (col_indices[i] == row)
            {
                diagonal[row] = values[i];
            }
        }
    }
}

__global__ void ExtractInverseDiagKernel(int n, const int* row_offsets, const int* col_indices, const double* values, double* diagonal) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n)
    {
        diagonal[row] = 0.0f; // Initialize diagonal element
        for(int i = row_offsets[row]; i < row_offsets[row + 1]; i++)
        {
            if (col_indices[i] == row)
            {
                diagonal[row] = 1 / values[i];
            }
        }
    }
}

// solves K\f  -> u
void CRS_Matrix_GPU::cg(std::vector<double> &u, std::vector<double> const &f, 
               int const max_iterations, double const eps) const
{
	assert(_ncols == _nrows);
    assert( _ncols == static_cast<int>(u.size()) ); // compatibility of inner dimensions
    assert( _nrows == static_cast<int>(f.size()) ); // compatibility of outer dimensions
    
    // allocate device memory for parameter list
    size_t nBytes = Nrows()*sizeof(u[0]);              // square matrix assumed
    Vec d_U(u);
    CHECK_CUDA( hipMemset(d_U.data(), 0x0, nBytes) )     // u = 0
    Vec d_F(f);
    
    // allocate device memory for aux vectors
    Vec d_R(Nrows());
    Vec d_W(Nrows()); 
    Vec d_S(Nrows());  
    Vec d_V(Nrows());  
       
    Vec d_D(Nrows());                              // i n v e r s e   of diagonal
    GetInvDiag(d_D);


    Defect(d_R,d_F,d_U);                               // r = f-K*u
    vdmult_gpu(d_W,d_R,d_D);                           // w = D^(-1)*r
    //d_W=d_R;                                         // no precond: w = r
    d_S=d_W;                                           // s = w
    
    double sigma;
    hipblasDdot(_cublasHandle, Nrows(), d_W.data(), 1, d_R.data(), 1, &sigma);   // sigmq = <w,r>
    double sigma0(sigma);
    int    iter(0);
    cout << iter << " iterations : error " << sqrt(sigma) << endl;

    while (sigma0*eps*eps<sigma && iter<max_iterations)
    {
		++iter;
		double sig_old(sigma);
		//Mult(v,sv = K*s
		Mult(d_V,d_S);                                 // v = K*s
		double denominator;
		hipblasDdot(_cublasHandle,  d_S.size(), d_S.data(), 1, d_V.data(), 1, &denominator); // <s,v>
		double alpha = sigma/denominator;              // alf = sig/<s,v>
                                                       // u = alf*s+u
		hipblasDaxpy(_cublasHandle, Nrows(), &alpha, d_S.data(), 1, d_U.data(), 1);
		double minus_alpha = -alpha;                   // r = -alf*v+r
		hipblasDaxpy(_cublasHandle, Nrows(), &minus_alpha, d_V.data(), 1, d_R.data(), 1);
		//                                             // w = D^(-1)*r
        //vdmult_elem<<<2*44,2*256>>>(Nrows(),d_W.data(),d_R.data(),d_D.data());
        vdmult_gpu(d_W,d_R,d_D);                       // w = D^(-1)*r   
        //d_W=d_r;                                       // no precond: w = r
		                                               // sig = <w,r>
		hipblasDdot(_cublasHandle, Nrows(), d_W.data(), 1, d_R.data(), 1, &sigma);
        //  cout << iter << " iterations : error " << sqrt(sigma) << "  rel. error: " << sqrt(sigma/sigma0) << endl;
		double beta = sigma/sig_old;
#define MYBLAS        
#ifndef MYBLAS
		//vdaxpy(s,w,beta,s);                       // s = w+beta*s
		// use d_V as aux vector; s --> v; w --> s; // s = beta*v+s    
        d_V = d_S;  //hipMemcpy(d_V.data(), d_S.data(), nBytes, hipMemcpyDeviceToDevice); // s-->v      
        d_S = d_W;  //hipMemcpy(d_S.data(), d_W.data(), nBytes, hipMemcpyDeviceToDevice); // w-->s
		hipblasDaxpy(_cublasHandle, Nrows(), &beta, d_V.data(), 1, d_S.data(), 1);
#else
        //vdxpay<<<2*44,2*256>>>(Nrows(), d_W.data(), beta, d_S.data()); // s = w+beta*s
        vdxpay_gpu(d_W, beta, d_S);                    // s = w+beta*s
#endif        
	}
	// transfer solution u from device to host.
	CopyDevice2Host(u, d_U);
	
    cout << iter << " iterations : error " << sqrt(sigma) << "  rel. error: " << sqrt(sigma/sigma0) << endl;    
}


//    profiling
//  nsys-ui  ./main.NVCC
//    or
//  nvvp ./main.NVCC_ data/square_100_4

void CRS_Matrix_GPU::JacobiSmoother(Vec const &f, Vec &u, Vec &r, int nsmooth, double omega, bool zero) const
{
    {
    // ToDO: ensure compatible dimensions
    assert(_ncols==_nrows);
    assert( _ncols == static_cast<int>(u.size()) ); // compatibility of inner dimensions
    assert( _nrows == static_cast<int>(r.size()) ); // compatibility of outer dimensions
    assert( r.size() == f.size() );
    
    Vec inv_diag(r.size());
    GetInvDiag(inv_diag);        // accumulated diagonal of matrix @p SK.
         
    if (zero) {            // assumes initial solution is zero
        for (int k = 0; k < _nrows; ++k) {
            // u := u + om*D^{-1}*f
            vdmult_gpu(u, inv_diag, f);
            hipblasDscal(_cublasHandle, _nrows, &omega, u.data(), 1);
        }
        --nsmooth;                           // first smoothing sweep done
    }

 
    for (int ns = 1; ns <= nsmooth; ++ns) {

        Defect(r, f, u);
        hipblasDaxpy(_cublasHandle, _nrows, &omega, r.data(), 1, u.data(), 1 );
    }
    return;
}
}