#include "hip/hip_runtime.h"
#include "matrix.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <sstream>
#include <hip/hip_runtime.h>

void print_first_5_elements( const float * vector )
{
    for( int i = 0; i < 5; i++ )
    {
        std::cout << vector[i] << ", ";
    }
    std::cout << " ..." << std::endl;
}

int main ()
{
    hipblasStatus_t  stat;
    hipblasHandle_t  handle;

    std::chrono::time_point<std::chrono::system_clock> t1, t2;
    std::chrono::duration<double, std::milli>  time_span;


    const int M = 100;      //can't be smaller than 5
    const int N = 100;

    stat = hipblasCreate (& handle );
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "ERROR: cublasInit() failed!" << std::endl;
        exit(1);
    }

    float *x_h = new float [N];
    float *y_h = new float [N];
    float *z_h = new float [N];
    float alpha = 2.0;
    float beta = 3.0;
    
    for (int i = 0; i < N; i++)
    {
        x_h[i] = (i % 11);
        y_h[i] = 11 - (i % 11);
        z_h[i] = 0;
    }

    float *x_d, *y_d, *z_d; // device data
    hipMalloc((void **) &x_d, N * sizeof(float));
    hipMalloc((void **) &y_d, N * sizeof(float));
    hipMalloc((void **) &z_d, N * sizeof(float));

    hipblasSetVector(N, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(N, sizeof(float), y_h, 1, y_d, 1);
    hipblasSetVector(N, sizeof(float), z_h, 1, z_d, 1);

    std::cout << "First five elements of the vectors x and y are: " << std::endl;
    std::cout << "x = ( ";
    print_first_5_elements(x_h);
    std::cout << "y = ( ";
    print_first_5_elements(y_h);
    std::cout << "After each calculation, they are reset to this state." << std::endl << std::endl;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSaxpy(handle, N, &alpha, x_d, 1, y_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(y_h, y_d, N * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "y = " << alpha <<"*x + y :" << std::endl << "y = ( ";
    print_first_5_elements(y_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //_______________________________________________________________________

    for (int i = 0; i < N; i++)
    {
        x_h[i] = (i % 11);
        y_h[i] = 11 - (i % 11);
    }
    hipblasSetVector(N, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(N, sizeof(float), y_h, 1, y_d, 1);
    const float dummy = 1;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSscal(handle, N, &alpha, x_d, 1);
    hipblasSaxpy( handle, N, &dummy, y_d, 1, x_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(x_h, x_d, N * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "x = " << alpha <<"*x + y :" << std::endl << "x = ( ";
    print_first_5_elements(x_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;

    //_________________________________________________________________________________
    
    for (int i = 0; i < N; i++)
    {
        x_h[i] = (i % 11);
        y_h[i] = 11 - (i % 11);
    }
    hipblasSetVector(N, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(N, sizeof(float), y_h, 1, y_d, 1);

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSscal(handle, N, &alpha, x_d, 1);
    hipblasSscal(handle, N, &beta, y_d, 1);
    hipblasSaxpy(handle, N, &dummy, x_d, 1, y_d, 1);
    hipblasSswap(handle, N, z_d, 1, y_d, 1);   
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(z_h, z_d, N * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "z = " << alpha <<"*x + " << beta << "*y :" << std::endl << "z = ( ";
    print_first_5_elements(z_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    for (int i = 0; i < N; i++)
    {
        x_h[i] = (i % 11);
        y_h[i] = 11 - (i % 11);
    }
    hipblasSetVector(N, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(N, sizeof(float), y_h, 1, y_d, 1);
    float dot_product = 0;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSdot(handle, N, x_d, 1, y_d, 1, &dot_product);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();

    time_span = t2 - t1;
    std::cout << "<x, y> :" << std::endl << "<x, y> = " << dot_product << std::endl;
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    for (int i = 0; i < N; i++)
    {
        x_h[i] = (i % 11);
    }
    hipblasSetVector(N, sizeof(float), x_h, 1, x_d, 1);
    float norm = 0;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSnrm2(handle, N, x_d, 1, &norm );
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();

    time_span = t2 - t1;
    std::cout << "||x|| :" << std::endl << "||x|| = " << norm << std::endl;
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    


    delete [] x_h;
    delete [] y_h;
    hipFree(x_d);
    hipFree(y_d);
    hipblasDestroy(handle );
}