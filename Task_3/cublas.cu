#include "hip/hip_runtime.h"
#include "matrix.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <sstream>
#include <hip/hip_runtime.h>
#include "equal_reduction.h"

void print_first_5_elements( const float * vector )
{
    for( int i = 0; i < 5; i++ )
    {
        std::cout << vector[i] << ", ";
    }
    std::cout << " ..." << std::endl;
}

void print_matrix( denseMatrix<float> M )
{
    std::cout << std::endl;
    for( int i = 0; i < M.GetNrows(); i++ )
    {
        for( int j = 0; j < M.GetNcols(); j++ )
        {
            std::cout << M[i + M.GetNrows() * j] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void print_matrix( float* M, int n, int m)
{
    std::cout << std::endl;
    for( int i = 0; i < n; i++ )
    {
        for( int j = 0; j < m; j++ )
        {
            std::cout << M[i + n * j] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}
int main ()
{
    hipblasStatus_t  stat;
    hipblasHandle_t  handle;

    std::chrono::time_point<std::chrono::system_clock> t1, t2;
    std::chrono::duration<double, std::milli>  time_span;

    const int n = 10000;
    const int m = 10000;      //can't be smaller than 5

    denseMatrix<float> M(n, m);

    std::cout << "n = " << n << std::endl << "m = " << m << std::endl << std::endl;

    //Setting up unit matrix with element (1, 2) equal 1 for testing.
    /*
    for( int i = 0; i < n; i++ )
    {
        for( int j = 0; j < m; j++ )
        {
            if( i == j )
            {
                M[j*n + i] = 1;
            }
            else
            {
                M[j*n + i] = 0;
            }
        }
    }
    M[1*n] = 1;
    */

    stat = hipblasCreate (& handle );
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "ERROR: cublasInit() failed!" << std::endl;
        exit(1);
    }

    float *x_h = new float [n];
    float *y_h = new float [n];
    float *z_h = new float [n];
    float alpha = 2.0;
    float beta = 3.0;
    
    for (int i = 0; i < n; i++)
    {
        x_h[i] = (i % 11);
        y_h[i] = 11 - (i % 11);
        z_h[i] = 0;
    }

    float *x_d, *y_d, *z_d; // device data
    hipMalloc((void **) &x_d, n * sizeof(float));
    hipMalloc((void **) &y_d, n * sizeof(float));
    hipMalloc((void **) &z_d, n * sizeof(float));

    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);
    hipblasSetVector(n, sizeof(float), z_h, 1, z_d, 1);

    std::cout << "First five elements of the vectors x and y are: " << std::endl;
    std::cout << "x = ( ";
    print_first_5_elements(x_h);
    std::cout << "y = ( ";
    print_first_5_elements(y_h);
    std::cout << "After each calculation in the first part, they are reset to this state." << std::endl << std::endl;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSaxpy(handle, n, &alpha, x_d, 1, y_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "y = " << alpha <<"*x + y :" << std::endl << "y = ( ";
    print_first_5_elements(y_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //_______________________________________________________________________

    for (int i = 0; i < n; i++)
    {
        y_h[i] = 11 - (i % 11);
    }
    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);
    const float one = 1;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSscal(handle, n, &alpha, x_d, 1);
    hipblasSaxpy( handle, n, &one, y_d, 1, x_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(x_h, x_d, n * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "x = " << alpha <<"*x + y :" << std::endl << "x = ( ";
    print_first_5_elements(x_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;

    //_________________________________________________________________________________
    
    for (int i = 0; i < n; i++)
    {
        x_h[i] = (i % 11);
    }
    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSscal(handle, n, &alpha, x_d, 1);
    hipblasSscal(handle, n, &beta, y_d, 1);
    hipblasSaxpy(handle, n, &one, x_d, 1, y_d, 1);
    hipblasSswap(handle, n, z_d, 1, y_d, 1);   
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(z_h, z_d, n * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "z = " << alpha <<"*x + " << beta << "*y :" << std::endl << "z = ( ";
    print_first_5_elements(z_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);
    float dot_product = 0;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSdot(handle, n, x_d, 1, y_d, 1, &dot_product);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();

    time_span = t2 - t1;
    std::cout << "<x, y> :" << std::endl << "<x, y> = " << dot_product << std::endl;
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    float norm = 0;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSnrm2(handle, n, x_d, 1, &norm );
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();

    time_span = t2 - t1;
    std::cout << "||x|| :" << std::endl << "||x|| = " << norm << std::endl;
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    std::cout << "-------------------------------------" << std::endl;

    delete [] x_h;
    hipFree(x_d);

    float* r_h = new float[n];
    x_h = new float[m];

    for (int i = 0; i < m; i++)
        x_h[i] = pow( -1, i );

    for( int i = 0; i < n; i++ )
        r_h[i] = 0;

    float* r_d;
    float* M_data_d;
    hipMalloc((void **) &r_d, n * sizeof(float));
    hipMalloc((void **) &x_d, m * sizeof(float));
    hipMalloc((void **) &M_data_d, m * n * sizeof(float));    

    hipblasSetVector(m, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), r_h, 1, r_d, 1);
    hipblasSetVector(m*n, sizeof(float), M.data(), 1, M_data_d, 1);

    float zero = 0.0f;
    hipblasSgemv(handle, HIPBLAS_OP_N, n, m, &one, M_data_d, n, x_d, 1, &zero, r_d, 1); //dummy call

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSgemv(handle, HIPBLAS_OP_N, n, m, &one, M_data_d, n, x_d, 1, &zero, r_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(r_h, r_d, n * sizeof(float), hipMemcpyDeviceToHost);

    //print_matrix(M);

    time_span = t2 - t1;
    std::cout << "r = M*x" << std::endl << "r = ( ";
    print_first_5_elements(r_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    delete [] x_h;
    hipFree(x_d);
    delete [] r_h;
    hipFree(r_d);

    r_h = new float[m];
    x_h = new float[n];

    for (int i = 0; i < n; i++)
        x_h[i] = 1;//pow( -1, i );

    for( int i = 0; i < m; i++ )
        r_h[i] = 0;

    hipMalloc((void **) &r_d, m * sizeof(float));
    hipMalloc((void **) &x_d, n * sizeof(float));

    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(m, sizeof(float), r_h, 1, r_d, 1);

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSgemv(handle, HIPBLAS_OP_T, n, m, &one, M_data_d, n, x_d, 1, &zero, r_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(r_h, r_d, m * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "r = trans(M)*x" << std::endl << "r = ( ";
    print_first_5_elements(r_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    std::cout << "-------------------------------------" << std::endl;

    delete [] x_h;
    hipFree(x_d);
    delete [] r_h;
    hipFree(r_d);


    float* T_h = new float[3*n];
    x_h = new float[n];
    r_h = new float[n];

    for( int i = 0; i < n; i++ )
    {
        T_h[3*i] = -1;  
        T_h[3*i + 1] = 2;
        T_h[3*i + 2] = -1;
    }
    T_h[0] = 0;
    T_h[3*n-1] = 0;

    for( int i = 0; i < n; i++)
        x_h[i] = 1;

    for( int i = 0; i < n; i++ )
        r_h[i] = 0;

    float* T_d;
    hipMalloc((void **) &T_d,3 * n * sizeof(float));
    hipMalloc((void **) &x_d, n * sizeof(float));
    hipMalloc((void **) &r_d, n * sizeof(float));

    hipblasSetVector(3 * n, sizeof(float), T_h, 1, T_d, 1);
    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), r_h, 1, r_d, 1);

    hipDeviceSynchronize();
    hipblasSgbmv(handle, HIPBLAS_OP_N, n, n, 1, 1, &one, T_d, 3, x_d, 1, &zero, r_d, 1); //dummy call

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSgbmv(handle, HIPBLAS_OP_N, n, n, 1, 1, &one, T_d, 3, x_d, 1, &zero, r_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(r_h, r_d, n * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "r = T*x" << std::endl << "r = ( ";
    print_first_5_elements(r_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
        
    //__________________________________________________________________________________
    
    std::cout << "-------------------------------------" << std::endl;

    delete [] x_h;
    hipFree(x_d);
    delete [] y_h;
    hipFree(y_d);
    delete [] z_h;
    hipFree(z_d);

    float* A_h = new float[n*n];
    x_h = new float[n];
    y_h = new float[n];
    z_h = new float[n];
    float* temp_h = new float[m];

    for( int i = 0; i < n; i++ )
    {
        x_h[i] = 1e-5;
        y_h[i] = 0;
        z_h[i] = 0;
    }

    float *A_d;
    float *temp_d;
    hipMalloc((void**) &A_d, n * n * sizeof(float) );
    hipMalloc((void**) &y_d, n * sizeof(float) );
    hipMalloc((void**) &z_d, n * sizeof(float) );
    hipMalloc((void**) &temp_d, n * sizeof(float) );

    hipblasSetVector( n, sizeof(float), x_h, 1, x_d, 1);

    //hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, m, &one, M_data_d, n, M_data_d, n, &zero, A_d, n ); //dummy call

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, m, &one, M_data_d, n, M_data_d, n, &zero, A_d, n );
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(A_h, A_d, n * n * sizeof(float), hipMemcpyDeviceToHost );

    time_span = t2 - t1;
    std::cout << "A = M*trans(M)" << std::endl;
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSgemv(handle, HIPBLAS_OP_N, n, n, &one, A_d, n, x_d, 1, &zero, z_d, 1 );
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(z_h, z_d, n * sizeof(float), hipMemcpyDeviceToHost );
    
    time_span = t2 - t1;
    std::cout << "z = A*x" << std::endl << "z = ( ";
    print_first_5_elements(z_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSgemv(handle, HIPBLAS_OP_T, n, m, &one, M_data_d, n, x_d, 1, &zero, temp_d, 1 );
    hipblasSgemv(handle, HIPBLAS_OP_N, n, m, &one, M_data_d, n, temp_d, 1, &zero, y_d, 1 );
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost );
    
    time_span = t2 - t1;
    std::cout << "y = M*(trans(M)*x)" << std::endl << "y = ( ";
    print_first_5_elements(y_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;

    float eps = 1e-3;
    bool equal = are_equal(y_d, z_d, n, eps );
    std::cout << "Vectors y and z are " << (equal ? "equal" : "not equal") << " with precision of" << eps << "." <<  std::endl << std::endl;

    //__________________________________________________________________________________

    std::cout << "-------------------------------------" << std::endl;

    if( n != m )
    {
        std::cout << "Can't extract diagonal from non-square matrix." << std::endl;
    }
    else
    {
        float* diag_h = new float[n];
        float* diag_d;
        hipMalloc( (void**) &diag_d, n * sizeof(float) );

        hipDeviceSynchronize();
        t1 = std::chrono::high_resolution_clock::now();
        hipblasScopy(handle, n, M_data_d, n + 1, diag_d, 1);
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        hipMemcpy(diag_h, diag_d, n * sizeof(float), hipMemcpyDeviceToHost );

        time_span = t2 - t1;
        std::cout << "y = diag(M)" << std::endl << "y = ( ";
        print_first_5_elements(diag_h);
        std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;

        delete [] diag_h;
        hipFree(diag_d);
    }


    //__________________________________________________________________________________

    delete [] x_h;
    delete [] y_h;
    delete [] z_h;
    delete [] r_h;
    delete [] T_h;
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipFree(r_d);
    hipFree(M_data_d);
    hipFree(T_d);
    hipblasDestroy(handle);
}