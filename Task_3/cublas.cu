#include "hip/hip_runtime.h"
#include "matrix.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <sstream>
#include <hip/hip_runtime.h>

void print_first_5_elements( const float * vector )
{
    for( int i = 0; i < 5; i++ )
    {
        std::cout << vector[i] << ", ";
    }
    std::cout << " ..." << std::endl;
}

void print_matrix( denseMatrix<float> M )
{
    std::cout << std::endl;
    for( int i = 0; i < M.GetNrows(); i++ )
    {
        for( int j = 0; j < M.GetNcols(); j++ )
        {
            std::cout << M[i + M.GetNrows() * j] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main ()
{
    hipblasStatus_t  stat;
    hipblasHandle_t  handle;

    std::chrono::time_point<std::chrono::system_clock> t1, t2;
    std::chrono::duration<double, std::milli>  time_span;

    const int n = 15;
    const int m = 10;      //can't be smaller than 5

    denseMatrix<float> M(n, m);

    stat = hipblasCreate (& handle );
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "ERROR: cublasInit() failed!" << std::endl;
        exit(1);
    }

    float *x_h = new float [n];
    float *y_h = new float [n];
    float *z_h = new float [n];
    float alpha = 2.0;
    float beta = 3.0;
    
    for (int i = 0; i < n; i++)
    {
        x_h[i] = (i % 11);
        y_h[i] = 11 - (i % 11);
        z_h[i] = 0;
    }

    float *x_d, *y_d, *z_d; // device data
    hipMalloc((void **) &x_d, n * sizeof(float));
    hipMalloc((void **) &y_d, n * sizeof(float));
    hipMalloc((void **) &z_d, n * sizeof(float));

    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);
    hipblasSetVector(n, sizeof(float), z_h, 1, z_d, 1);

    std::cout << "First five elements of the vectors x and y are: " << std::endl;
    std::cout << "x = ( ";
    print_first_5_elements(x_h);
    std::cout << "y = ( ";
    print_first_5_elements(y_h);
    std::cout << "After each calculation, they are reset to this state." << std::endl << std::endl;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSaxpy(handle, n, &alpha, x_d, 1, y_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "y = " << alpha <<"*x + y :" << std::endl << "y = ( ";
    print_first_5_elements(y_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //_______________________________________________________________________

    for (int i = 0; i < n; i++)
    {
        y_h[i] = 11 - (i % 11);
    }
    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);
    const float one = 1;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSscal(handle, n, &alpha, x_d, 1);
    hipblasSaxpy( handle, n, &one, y_d, 1, x_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(x_h, x_d, n * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "x = " << alpha <<"*x + y :" << std::endl << "x = ( ";
    print_first_5_elements(x_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;

    //_________________________________________________________________________________
    
    for (int i = 0; i < n; i++)
    {
        x_h[i] = (i % 11);
    }
    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSscal(handle, n, &alpha, x_d, 1);
    hipblasSscal(handle, n, &beta, y_d, 1);
    hipblasSaxpy(handle, n, &one, x_d, 1, y_d, 1);
    hipblasSswap(handle, n, z_d, 1, y_d, 1);   
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(z_h, z_d, n * sizeof(float), hipMemcpyDeviceToHost);

    time_span = t2 - t1;
    std::cout << "z = " << alpha <<"*x + " << beta << "*y :" << std::endl << "z = ( ";
    print_first_5_elements(z_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    hipblasSetVector(n, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), y_h, 1, y_d, 1);
    float dot_product = 0;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSdot(handle, n, x_d, 1, y_d, 1, &dot_product);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();

    time_span = t2 - t1;
    std::cout << "<x, y> :" << std::endl << "<x, y> = " << dot_product << std::endl;
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    float norm = 0;

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSnrm2(handle, n, x_d, 1, &norm );
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();

    time_span = t2 - t1;
    std::cout << "||x|| :" << std::endl << "||x|| = " << norm << std::endl;
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;
    
    //__________________________________________________________________________________

    delete [] x_h;
    hipFree(x_d);


    float* r_h = new float[n];
    x_h = new float[m];

    for (int i = 0; i < m; i++)
        x_h[i] = 1;

    for( int i = 0; i < n; i++ )
        r_h[i] = 0;

    float* r_d;
    hipMalloc((void **) &r_d, n * sizeof(float));
    hipMalloc((void **) &x_d, m * sizeof(float));
    

    hipblasSetVector(m, sizeof(float), x_h, 1, x_d, 1);
    hipblasSetVector(n, sizeof(float), r_h, 1, r_d, 1);

    float zero = 0.0f;
    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    hipblasSgemv(handle, HIPBLAS_OP_N, n, m, &one, M.data(), n, x_d, 1, &zero, r_d, 1);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(r_h, r_d, n * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Error: " << hipGetErrorString( hipGetLastError() )<< std::endl;
    print_matrix(M);

    time_span = t2 - t1;
    std::cout << "r = M*x" << std::endl << "r = ( ";
    print_first_5_elements(r_h);
    std::cout << "Time: " << time_span.count() << " milliseconds." << std::endl << std::endl;






    delete [] x_h;
    delete [] y_h;
    delete [] z_h;
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipblasDestroy(handle );
}