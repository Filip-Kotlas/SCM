#include "hip/hip_runtime.h"
//Filip Kotlas
//task 2 subtask 4

// originates from Ruetsch/Oster: Getting Started with CUDA
// more C++-style by Haase
#include <cassert>
#include <iostream>
#include <cmath>
#include <iomanip>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/system/cuda/execution_policy.h>

struct increment
{
    __host__ __device__
    float operator()(const float &x)
    {
        return x + 1;
    }
};

struct logarithm
{
    __host__ __device__
    float operator()(const float &x)
    {
        return log(x);
    }
};

struct exponential
{
    __host__ __device__
    float operator()(const float &x)
    {
        return exp(x);
    }
};

using namespace std;

int main(void)
{
    cout << setprecision(10);
    int const N = 10000;
    int const nBytes = N * sizeof(float);

    int const blockSize = 64;
    int const numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_span = 0;

    float *a, *b;                    // device data
    hipMallocManaged(&a, nBytes);
    hipMallocManaged(&b, nBytes);

    thrust::sequence(a, a + N, 100.0f, 1.0f );

    hipMemcpy(b, a, nBytes, hipMemcpyDeviceToDevice);  //  b <- a

// ---------------------------------------------------------
// Manipulate on GPU
    hipEventRecord(start);
    thrust::transform(thrust::cuda::par, b, b + N, b, increment());             //  b := b+1.0
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    cout << hipGetErrorName(hipGetLastError()) << endl;
    
// Check on CPU
    for (int i = 0; i < N; i++) assert( a[i] == b[i] - 1.0f );
    cout << "Check 1  OK" << endl;
    hipEventElapsedTime(&time_span, start, stop);
    cout << "Incrementing: The ellapsed time is " << time_span << " milliseconds." << endl;
// ---------------------------------------------------------

    hipFree(a);
    hipFree(b);
    cout << endl;

//_______________________________________________________________
// My extension of the code
    
    //Task 2, subtask 1
   
    float *summand_1, *summand_2, *result;
    hipMallocManaged(&summand_1, nBytes);
    hipMallocManaged(&summand_2, nBytes);
    hipMallocManaged(&result, nBytes);

    thrust::sequence(summand_1, summand_1 + N, 0.0f, 1.0f);
    thrust::sequence(summand_2, summand_2 + N, static_cast<float>(N), -1.0f);

    hipEventRecord(start);
    thrust::transform(thrust::cuda::par, summand_1, summand_1 + N, summand_2, result, thrust::plus<float>() );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    cout << hipGetErrorName(hipGetLastError()) << endl;

    bool success = true;
    for( int i = 0; i < N; i++)
    {
        if( result[i] != N )
        {
            cout << "Addition: Instead of " << N << " there is " << result[i] << " on position " << i << "." << endl;
            success = false;
        }
    }
    if(success)
        cout << "Vectors were successfuly added together." << endl;

    hipEventElapsedTime(&time_span, start, stop);
    cout << "Summation: The ellapsed time is " << time_span << " milliseconds." << endl;

    hipFree(summand_1);
    hipFree(summand_2);
    hipFree(result);
    cout << endl;

    //Task 2, subtask 3
    float *argument, *after_logarithm, *after_exponentiation;
    hipMallocManaged(&argument, nBytes);
    hipMallocManaged(&after_logarithm, nBytes);
    hipMallocManaged(&after_exponentiation, nBytes);

    thrust::sequence(argument, argument + N, 0.0f, 1.0f );

    hipEventRecord(start);
    thrust::transform(thrust::cuda::par, argument, argument + N, after_logarithm, logarithm());
    thrust::transform(thrust::cuda::par, after_logarithm, after_logarithm + N, after_exponentiation, exponential());
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    cout << hipGetErrorName(hipGetLastError()) << endl;

    success = true;
    float precision = 1e-1;
    for( int i = 0; i < N; i++)
    {
        if( abs(after_exponentiation[i] - argument[i]) > precision )
        {
            cout << abs(after_exponentiation[i] - argument[i]) << endl;
            cout << "Log & exp: Instead of " << argument[i] << " we get " << after_exponentiation[i] << " on position " << i << "." << endl;
            success = false;
        }
    }
    if(success)
        cout << "Log and exp where successfully applied to the vector. The results agree with an accuracy of " << precision << "." << endl;

    hipEventElapsedTime(&time_span, start, stop);
    cout << "Log & exp: The ellapsed time is " << time_span << " milliseconds." << endl;

    hipFree(argument);
    hipFree(after_logarithm);
    hipFree(after_exponentiation);
    cout << endl;

    return 0;
}
