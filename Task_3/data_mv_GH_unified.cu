//Filip Kotlas
//task 2 subtask 4

// originates from Ruetsch/Oster: Getting Started with CUDA
// more C++-style by Haase

#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <cmath>
#include <iomanip>

__global__ void inc_gpu(float *const a, int N);

__global__ void inc_gpu(float *const a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        a[idx] = a[idx] + 1;
}

__global__ void sum_gpu(float *const a, float *const b, float* const c, int N);
__global__ void sum_gpu(float *const a, float *const b, float* const c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N )
    {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void log_gpu(float* arg, float* res, int N);
__global__ void log_gpu(float* arg, float* res, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N )
    {
        res[idx] = log(arg[idx]);
    }
}

__global__ void exp_gpu(float* arg, float* res, int N);
__global__ void exp_gpu(float* arg, float* res, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N )
    {
        res[idx] = exp(arg[idx]);
    }
}

using namespace std;

int main(void)
{
    cout << setprecision(10);
    int const N = 10000;
    int const nBytes = N * sizeof(float);

    int const blockSize = 64;
    int const numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_span = 0;

    float *a, *b;                    // device data
    hipMallocManaged(&a, nBytes);
    hipMallocManaged(&b, nBytes);

    for (int i = 0; i < N; i++) a[i] = 100.0f + static_cast<float>(i);

    hipMemcpy(b, a, nBytes, hipMemcpyDeviceToDevice);  //  b <- a

// ---------------------------------------------------------
// Manipulate on GPU
    hipEventRecord(start);
    inc_gpu <<< numBlocks, blockSize>>>(b, N);             //  b := b+1.0
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    cout << hipGetErrorName(hipGetLastError()) << endl;
    
// Check on CPU
    for (int i = 0; i < N; i++) assert( a[i] == b[i] - 1.0f );
    cout << "Check 1  OK" << endl;
    hipEventElapsedTime(&time_span, start, stop);
    cout << "Incrementing: The ellapsed time is " << time_span << " milliseconds." << endl;
// ---------------------------------------------------------

    hipFree(a);
    hipFree(b);
    cout << endl;

//_______________________________________________________________
// My extension of the code
    
    //Task 2, subtask 1
   
    float *summand_1, *summand_2, *result;
    hipMallocManaged(&summand_1, nBytes);
    hipMallocManaged(&summand_2, nBytes);
    hipMallocManaged(&result, nBytes);

    for( int i = 0; i < N; i++ )
    {
        summand_1[i] = static_cast<float>(i);
        summand_2[i] = static_cast<float>(N) - static_cast<float>(i);
    }

    hipEventRecord(start);
    sum_gpu <<< numBlocks, blockSize >>> (summand_1, summand_2, result, N );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    cout << hipGetErrorName(hipGetLastError()) << endl;

    bool success = true;
    for( int i = 0; i < N; i++)
    {
        if( result[i] != N )
        {
            cout << "Addition: Instead of " << N << " there is " << result[i] << " on position " << i << "." << endl;
            success = false;
        }
    }
    if(success)
        cout << "Vectors were successfuly added together." << endl;

    hipEventElapsedTime(&time_span, start, stop);
    cout << "Summation: The ellapsed time is " << time_span << " milliseconds." << endl;

    hipFree(summand_1);
    hipFree(summand_2);
    hipFree(result);
    cout << endl;

    //Task 2, subtask 3
    float *argument, *after_logarithm, *after_exponentiation;
    hipMallocManaged(&argument, nBytes);
    hipMallocManaged(&after_logarithm, nBytes);
    hipMallocManaged(&after_exponentiation, nBytes);

    for( int i = 0; i < N; i++ )
    {
        argument[i] = static_cast<float>(i);
    }

    hipEventRecord(start);
    log_gpu <<< numBlocks, blockSize >>> (argument, after_logarithm, N );
    exp_gpu <<< numBlocks, blockSize >>> (after_logarithm, after_exponentiation, N );
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    cout << hipGetErrorName(hipGetLastError()) << endl;

    success = true;
    float precision = 1e-1;
    for( int i = 0; i < N; i++)
    {
        if( abs(after_exponentiation[i] - argument[i]) > precision )
        {
            cout << abs(after_exponentiation[i] - argument[i]) << endl;
            cout << "Log & exp: Instead of " << argument[i] << " we get " << after_exponentiation[i] << " on position " << i << "." << endl;
            success = false;
        }
    }
    if(success)
        cout << "Log and exp where successfully applied to the vector. The results agree with an accuracy of " << precision << "." << endl;

    hipEventElapsedTime(&time_span, start, stop);
    cout << "Log & exp: The ellapsed time is " << time_span << " milliseconds." << endl;

    hipFree(argument);
    hipFree(after_logarithm);
    hipFree(after_exponentiation);
    cout << endl;

    return 0;
}
